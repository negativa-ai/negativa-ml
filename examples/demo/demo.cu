#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <sstream>

#include <hip/hip_runtime.h>

#include "demo.h"

__global__ void matrixMulGPU(int *a, int *b, int *c, int m, int n, int p)
{
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling,
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   *
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN YOUR SOLUTION
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = gridDim.x * blockDim.x;
  size_t size = m * p;
  for (; gid < size; gid += stride)
  {
    c[gid] = 0;
    size_t i = gid / p;
    size_t j = gid % p;
    for (int k = 0; k < n; k++)
    {
      int a_ik = a[i * n + k];
      int b_kj = b[k * p + j];
      c[gid] += a_ik * b_kj;
    }
  }
  /// END YOUR SOLUTION
}

void matMul(int *a, int *b, int *c, int m, int n, int p)
{

  int size_a = m * n * sizeof(int); // Number of bytes of an M x N matrix
  int size_b = n * p * sizeof(int); // Number of bytes of an N x P matrix
  int size_c = m * p * sizeof(int); // Number of bytes of an M x P matrix

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size_a);
  hipMalloc(&d_b, size_b);
  hipMalloc(&d_c, size_c);

  hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */

  dim3 threads_per_block = 1024;
  dim3 number_of_blocks = 32;

  matrixMulGPU<<<number_of_blocks, threads_per_block, 0>>>(d_a, d_b, d_c, m, n, p);

  hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  // Free all our allocated memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

__global__ void setScalarItemGPU(int scalar, int *output, int height, int width)
{

  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = gridDim.x * blockDim.x;
  size_t size = height * width;
  for (; gid < size; gid += stride)
  {
    output[gid] = scalar;
  }
}

void setScalarItems(int scalar, int *output, int height, int width)
{
  int size_output = height * width * sizeof(int);

  int *d_output;
  hipMalloc(&d_output, size_output);

  dim3 threads_per_block = 1024;
  dim3 number_of_blocks = 32;

  setScalarItemGPU<<<number_of_blocks, threads_per_block, 0>>>(scalar, d_output, height, width);

  hipMemcpy(output, d_output, size_output, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  // Free all our allocated memory
  hipFree(d_output);
}